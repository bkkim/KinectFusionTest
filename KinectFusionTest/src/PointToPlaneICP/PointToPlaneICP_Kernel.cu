#include "hip/hip_runtime.h"

#include "cuda_SimpleMatrixUtil.h"
#include "PointToPlaneICP_Params.h"
#include "PointToPlaneICP_Data.h"

#define T_PER_BLOCK		16

/////////////////////////////////////////////////////////////////////////
// For correspondence check

__global__ 
void compute_correspondences(float4*  _frame_vertex, 
	                         float4*  _frame_normal, 
	                         float4*  _model_vertex, 
	                         float4*  _model_normal,
	                         float4*  _corre_vertex, 
	                         float4*  _corre_normal, 
	                         int      _width, 
	                         int      _height,
	                         float    _fx, 
	                         float    _fy, 
	                         float    _cx, 
	                         float    _cy,
	                         float    _z_min, 
	                         float    _z_max, 
	                         float    _thres_corres_dist, 
	                         float    _thres_corres_normal,
	                         float4x4 _delta_transform)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= _width || y >= _height) return;

	_corre_vertex[y*_width + x] = make_float4(MINF, MINF, MINF, MINF);
	_corre_normal[y*_width + x] = make_float4(MINF, MINF, MINF, MINF);

	float4 pFv = _frame_vertex[y*_width + x];
	float4 pFn = _frame_normal[y*_width + x];
	float4 pFc = make_float4(MINF, MINF, MINF, MINF);

	if (pFv.x != MINF && pFn.x != MINF)
	{
		float4 pTv = _delta_transform * pFv; // it is a vertex
		float4 pTn = _delta_transform * pFn; // it is a normal
	
		uint u = (uint)(((pTv.x * _fx) / pTv.z) + _cx);
		uint v = (uint)(((pTv.y * _fy) / pTv.z) + _cy);
	
		if (u >= 0 && u < _width && v >= 0 && v < _height)
		{
			unsigned int idx = v * _width + u;
			float4 pMv = _model_vertex[idx];
			float4 pMn = _model_normal[idx];
	
			if (pMv.x != MINF && pMn.x != MINF)
			{
				float dist_vertex = length(pTv - pMv);
				float dist_normal = dot(pTn, pMn);
	
				if (dist_vertex <= _thres_corres_dist && dist_normal >= _thres_corres_normal)
				{
					float w = max(0.0, 0.5f*((1.0f - dist_vertex / _thres_corres_dist) + (1.0f - ((pTv.z - _z_min) / (_z_max - _z_min)))));	// for weighted ICP
					
					_corre_vertex[y*_width + x] = pMv;
					_corre_normal[y*_width + x] = make_float4(pMn.x, pMn.y, pMn.z, w);
				}
			}
		}
	}
}

extern "C" void
launch_compute_correspondences(PointToPlaneICP_Data&   icp_data,
	                           PointToPlaneICP_Params& icp_params,
	                           float4x4&               delta_transform)
{
	const dim3 gridSize((icp_params.width + T_PER_BLOCK - 1) / T_PER_BLOCK, (icp_params.height + T_PER_BLOCK - 1) / T_PER_BLOCK);
	const dim3 blockSize(T_PER_BLOCK, T_PER_BLOCK);

	compute_correspondences <<< gridSize, blockSize >>> (icp_data.d_frame_vertex,
		                                                 icp_data.d_frame_normal,
		                                                 icp_data.d_model_vertex,
		                                                 icp_data.d_model_normal,
		                                                 icp_data.d_corre_vertex,
		                                                 icp_data.d_corre_normal,
		                                                 icp_params.width,
		                                                 icp_params.height,
		                                                 icp_params.fx,
		                                                 icp_params.fy,
		                                                 icp_params.cx,
		                                                 icp_params.cy,
		                                                 icp_params.z_min,
		                                                 icp_params.z_max,
		                                                 icp_params.thres_corres_dist,
		                                                 icp_params.thres_corres_normal,
		                                                 delta_transform);
#ifdef _DEBUG
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError(__FUNCTION__);
#endif

}
